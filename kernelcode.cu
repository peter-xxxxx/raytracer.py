
#include <hip/hip_runtime.h>
#include "math.h"
#include "float.h"

__device__ void normalize_vector(float *a, float *b, float *c)
{
    float len;
    float ta, tb, tc;

    ta = *a;
    tb = *b;
    tc = *c;
    len = ta*ta + tb*tb + tc*tc;

    if(len == 0.0)
        return;
    len = sqrt(len);
    *a = ta/len;
    *b = tb/len;
    *c = tc/len;
}

__device__ void intersectGPU(float *ray_array, float *scene, float *output, int *hit_obj_index, int scene_size)
{

/*
input: ray_array, scene, scene_size
output: [hit_point v3, hit_normal v3]
*/

    int i;
    int scene_index;
    int flag;
    int current_obj_i = -1;

    float l1, l2, l3;
    float t_ca;
    float d_squared;
    float radius_squared;
    float t_hc;
    float t;
    float hit_point1;
    float hit_point2;
    float hit_point3;
    float hit_normal1;
    float hit_normal2;
    float hit_normal3;
    float current_t = FLT_MAX;

    for(i = 0; i < scene_size; i++){
        flag = 0;
        scene_index = i*16;
        if (scene[scene_index] < 0.5){ /* means sphere */
            /* calculate intersect */
            l1 = scene[scene_index+1] - ray_array[0];
            l2 = scene[scene_index+2] - ray_array[1];
            l3 = scene[scene_index+3] - ray_array[2];
            t_ca = l1*ray_array[3] +
                   l2*ray_array[4] +
                   l3*ray_array[5];
            if(t_ca < 0)
                flag = 1;
            d_squared = l1*l1 + l2*l2 +l3*l3 - t_ca*t_ca;
            radius_squared = scene[scene_index+4]*scene[scene_index+4];
            if(d_squared > radius_squared)
                flag = 1;
            t_hc = sqrt(radius_squared - d_squared);
            t = t_ca - t_hc;
            if(t < 0)
                t = t_ca + t_hc;
            hit_point1 = ray_array[0] + t * ray_array[3];
            hit_point2 = ray_array[1] + t * ray_array[4];
            hit_point3 = ray_array[2] + t * ray_array[5];
            hit_normal1 = hit_point1 - scene[scene_index+1];
            hit_normal2 = hit_point2 - scene[scene_index+2];
            hit_normal3 = hit_point3 - scene[scene_index+3];
            normalize_vector(&hit_normal1, &hit_normal2, &hit_normal3);
        }
        if(flag == 0){
            if(t < current_t){
                current_t = t;
                current_obj_i = i;
                output[0] = hit_point1;
                output[1] = hit_point2;
                output[2] = hit_point3;
                output[3] = hit_normal1;
                output[4] = hit_normal2;
                output[5] = hit_normal3;
            }
        }
    }
    *hit_obj_index = current_obj_i;
}

__device__ void trace_non_diffuseGPU (float *ray, int hit_obj_index, float *hit_point, float *hit_normal,
                                      float *scene, int scene_size, int depth, int max_depth, float *output);
__device__ void trace_diffuseGPU(int obj_index, float *hit_point, float *hit_normal, float *scene, int scene_size, float *output);

__device__ void trace_recursivelyGPU(float *ray, float *scene, int scene_size, float *output, int depth, int max_depth){
    /*
    hit_object, hit_point, hit_normal = self.__intersect(ray)
    if hit_object is None:
        return Vector3(0.3, 0.3, 0.3)  # horizon
    traced_color = Vector3()
    if not hit_object.material.is_diffuse and depth < self.__max_recursion_depth:
        traced_color = self.__trace_non_diffuse(ray, hit_object, hit_point, hit_normal, depth)
    else:
        traced_color = self.__trace_diffuse(hit_object, hit_point, hit_normal)
    return traced_color + hit_object.material.emission_color
    */

    int hit_obj_index;
    float output_intersect[6];

    intersectGPU(ray, scene, output_intersect, &hit_obj_index, scene_size);

    if (hit_obj_index < 0) {
        output[0] = 0.3;
        output[1] = 0.3;
        output[2] = 0.3;
        return;
    }

    float traced_color[3];

    if (depth < max_depth && scene[16*hit_obj_index+14] < 0.5)
        /* trace_non_diffuseGPU (ray, hit_obj_index, output_intersect, output_intersect + 3,
                              scene, scene_size, depth, max_depth, traced_color);*/
        trace_diffuseGPU (hit_obj_index, output_intersect, output_intersect + 3, scene, scene_size, traced_color);
    else
        trace_diffuseGPU (hit_obj_index, output_intersect, output_intersect + 3, scene, scene_size, traced_color);

    traced_color[0] += scene[16*hit_obj_index + 8];
    traced_color[1] += scene[16*hit_obj_index + 9];
    traced_color[2] += scene[16*hit_obj_index + 10];

    output[0] = traced_color[0];
    output[1] = traced_color[1];
    output[2] = traced_color[2];

}

__device__ void trace_diffuseGPU(int obj_index, float *hit_point, float *hit_normal, float *scene, int scene_size, float *output){
    /*
    summed_color = Vector3()
    for light in filter(lambda obj: obj.is_light, self.__scene):
        transmission = Vector3(1, 1, 1)
        light_direction = (light.primitive.position - hit_point).normalize()
        for other in filter(lambda obj: obj != light, self.__scene):
            if other.primitive.intersect(Ray(hit_point + self.__bias * hit_normal,
                                         light_direction)):
                transmission = Vector3()
                break
        summed_color = summed_color + (
            hit_object.material.surface_color
            .mul_comp(transmission)
            .mul_comp(light.material.emission_color) *
            max(0, hit_normal.dot(light_direction)))
    return summed_color
    */

    int i;
    float summed_color[3] = {0, 0, 0};
    float transmission[3] = {1, 1, 1};
    int hit_obj_index;

    float output_intersect[6];
    float current_ray[7];
    float temp;


    for(i = 0; i < scene_size; i++){
        if(scene[i*16 + 15] > 0.5){ /* is light */


            current_ray[0] = hit_point[0] + 0.0001*hit_normal[0];
            current_ray[1] = hit_point[1] + 0.0001*hit_normal[1];
            current_ray[2] = hit_point[2] + 0.0001*hit_normal[2];

            current_ray[3] = scene[i*16 + 1] - hit_point[0];
            current_ray[4] = scene[i*16 + 2] - hit_point[1];
            current_ray[5] = scene[i*16 + 3] - hit_point[2];

            current_ray[6] = 1.0;

            normalize_vector(current_ray+3, current_ray+4, current_ray+5);

            for(i = 0; i < scene_size; i++){
                if(scene[i*16 + 15] < 0.5)
                    intersectGPU(current_ray, scene+i, output_intersect, &hit_obj_index, 1);

                if(hit_obj_index < 0){
                    transmission[0] = 0;
                    transmission[1] = 0;
                    transmission[2] = 0;
                    break;
                }
            }

            temp = hit_normal[0]*current_ray[3] + hit_normal[1]*current_ray[4] + hit_normal[2]*current_ray[5];
            temp = 0 > temp ? 0 : temp;

            summed_color[0] += scene[obj_index*16 + 5] * transmission[0] * scene[i*16 + 8] * temp;
            summed_color[1] += scene[obj_index*16 + 6] * transmission[1] * scene[i*16 + 9] * temp;
            summed_color[2] += scene[obj_index*16 + 7] * transmission[2] * scene[i*16 + 10] * temp;
        }

    }
    output[0] = summed_color[0];
    output[1] = summed_color[1];
    output[2] = summed_color[2];

}

__device__ void trace_non_diffuseGPU (float *ray, int hit_obj_index, float *hit_point, float *hit_normal,
                                      float *scene, int scene_size, int depth, int max_depth, float *output)
{
    /*
    inside = ray.direction.dot(hit_normal) > 0
    if inside:
        hit_normal = -hit_normal
    facing_ratio = -ray.direction.dot(hit_normal)
    fresnel = self.__mix((1 - facing_ratio) ** 2, 1, 0.1)
    reflection_ray = Ray(hit_point + self.__bias * hit_normal,
                         ray.direction.reflect(hit_normal).normalize())
    reflection = self.__trace_recursively(reflection_ray, depth + 1)
    refraction = Vector3()

    # transparent?
    if hit_object.material.transparency > 0:
        from_ior = ray.current_ior if inside else hit_object.material.ior
        to_ior = hit_object.material.ior if inside else ray.current_ior
        refraction_ray = Ray(hit_point - self.__bias * hit_normal,
                             ray.direction.refract(from_ior, to_ior, hit_normal)
                             .normalize())
        refraction = self.__trace_recursively(refraction_ray, depth + 1)

    # mix according to fresnel
    return ((reflection * fresnel +
            refraction * (1 - fresnel) * hit_object.material.transparency)
            .mul_comp(hit_object.material.surface_color))
    */

    if (ray[3]*hit_normal[0] + ray[4]*hit_normal[1] + ray[5]*hit_normal[2] > 0){
        hit_normal[0] *= -1;
        hit_normal[1] *= -1;
        hit_normal[2] *= -1;
    }
    float facing_ratio = - (ray[3]*hit_normal[0] + ray[4]*hit_normal[1] + ray[5]*hit_normal[2]);
    float fresnel = (1 - facing_ratio) * (1 - facing_ratio) * 0.9 + 0.1;

    float reflection_ray[7];

    reflection_ray[0] = hit_point[0] - 0.0001*hit_normal[0];
    reflection_ray[1] = hit_point[1] - 0.0001*hit_normal[1];
    reflection_ray[2] = hit_point[2] - 0.0001*hit_normal[2];

    float temp;
    temp = ray[0]*hit_normal[0] + ray[1]*hit_normal[1] + ray[2]*hit_normal[2];
    temp *= 2;

    reflection_ray[3] = ray[0] - temp * hit_normal[0];
    reflection_ray[4] = ray[1] - temp * hit_normal[1];
    reflection_ray[5] = ray[2] - temp * hit_normal[2];

    normalize_vector(reflection_ray+3, reflection_ray+4, reflection_ray+5);

    reflection_ray[6] = 1.0;

    float reflection[3];
    float refraction[3] = {0, 0, 0};

    trace_recursivelyGPU(reflection_ray, scene, scene_size, reflection, depth+1, max_depth);

    /*
    def refract(self, from_ior, to_ior, normal):
        # Refracts the vector with regard to material change and normal

        eta = to_ior / from_ior
        cos_i = -normal.dot(self)
        k = 1 - eta ** 2 * (1 - cos_i ** 2)
        return self * eta + normal * (eta * cos_i - math.sqrt(k))
    */

    if (scene[hit_obj_index*16 + 12] > 0){
        float from_ior, to_ior;
        if (ray[3]*hit_normal[0] + ray[4]*hit_normal[1] + ray[5]*hit_normal[2] > 0) {
            from_ior = ray[6];
            to_ior = scene[hit_obj_index*16 + 13];
        }
        else {
            from_ior = scene[hit_obj_index*16 + 13];
            to_ior = ray[6];
        }

        float refraction_ray[7];

        refraction_ray[0] = hit_point[0] - 0.0001*hit_normal[0];
        refraction_ray[1] = hit_point[1] - 0.0001*hit_normal[1];
        refraction_ray[2] = hit_point[2] - 0.0001*hit_normal[2];

        float eta = to_ior / from_ior;
        float cos_i = -(ray[3]*hit_normal[0] + ray[4]*hit_normal[1] + ray[5]*hit_normal[2]);
        float k = 1 - eta*eta * (1 - cos_i*cos_i);

        temp = eta*cos_i - sqrt(k);

        refraction_ray[3] = ray[3] * eta + hit_normal[0] * temp;
        refraction_ray[4] = ray[4] * eta + hit_normal[1] * temp;
        refraction_ray[5] = ray[5] * eta + hit_normal[2] * temp;

        normalize_vector(refraction_ray+3, refraction_ray+4, refraction_ray+5);

        refraction_ray[6] = 1;



        trace_recursivelyGPU(refraction_ray, scene, scene_size, refraction, depth+1, max_depth);

    }

    output[0] = (reflection[0] * fresnel + refraction[0] * (1-fresnel) * scene[hit_obj_index*16 + 12]) * scene[hit_obj_index*16 + 5];
    output[1] = (reflection[1] * fresnel + refraction[1] * (1-fresnel) * scene[hit_obj_index*16 + 12]) * scene[hit_obj_index*16 + 6];
    output[2] = (reflection[2] * fresnel + refraction[2] * (1-fresnel) * scene[hit_obj_index*16 + 12]) * scene[hit_obj_index*16 + 7];



}


__global__ void traceGPU(float *ray_array, float *scene, int width, int scene_size, float *output, int max_depth){

    int tx;
    int ray_array_index;

    tx = blockIdx.x*blockDim.x + threadIdx.x;


    if (tx >= width)
        return;

    ray_array_index = tx;

    int hit_obj_index;
    float output_c[3];



    trace_recursivelyGPU(ray_array + ray_array_index*7, scene, scene_size, output_c, 0, max_depth);

    output[0] = output_c[0];
    output[1] = output_c[1];
    output[2] = output_c[2];

}
